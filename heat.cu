#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<omp.h>
#include<math.h>

#define N 64
#define ITER_MAX 1000

int main(void){

    double tol = 1e-9;
    double error = tol + 1e-9;

    double * restrict A = malloc(N * N * sizeof(double));
    double * restrict Anew = malloc(N * N * sizeof(double));
	
	int x = 0;
	int radStart = floor((N-1)*0.3), radEnd = ceil((N-1)*0.7);

	int t0, t1;

	int pointx = floor((N-1)*0.5), pointy = floor((N-1)*0.5);
	
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
            int ij = i * N + j;
			Anew[ij] = 10;
            A[ij] = 10;
		}
	}

	for(int i = radStart; i <= radEnd; i++){
		Anew[i * N + (N-1)] = 100;
		A[i * N + (N-1)] = 100;
	}

    int iter = 0;

    double s_time = omp_get_wtime();
    
	while(error > tol && iter < ITER_MAX){
        error = 0.0;
		for(int i = 1; i < N-1; i++){
			for(int j = 1; j < N-1; j++){
                int ij = i * N + j;
                int ipj = (i + 1) * N + j;
                int imj = (i - 1) * N + j;
                int ijp = i * N + (j + 1);
                int ijm = i * N + (j - 1);
				Anew[ij] = 0.25 * (A[imj] + A[ipj] + A[ijm] + A[ijp]);
                error = fmax(error, fabs(A[ij] - Anew[ij]));
			}
		}
        for(int i = 0; i < N - 1; i++){
            for(int j = 0; j < N - 1; j++){
                int ij = i * N + j;
                A[ij] = Anew[ij];
            }
        }
        iter++;
    }


    double e_time = omp_get_wtime();
    
    printf("Time took %lf seconds", e_time - s_time); 
    

}
    


